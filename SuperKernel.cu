#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Kernels/AddSleep.cu"


__device__ JobPointer executeJob(volatile JobPointer currentJob);

__global__ void superKernel(volatile Queue incoming, 
                            volatile Queue results, volatile int *kill)
{ 
    // init and result are arrays of integers where result should end up
    // being the result of incrementing all elements of init.
    // They have n elements and are (n+1) long. The should wait for the
    // first element to be set to zero
    int warp_size = 32;

    int threadID = threadIdx.x % warp_size;
    int warpID = threadIdx.x / warp_size;   //added depenency on block

    __shared__ JobPointer currentJobs[32];

    while(!(*kill))
    {
      if(threadID==0)
          FrontAndDequeueJob(incoming, &currentJobs[warpID], kill);
      if(*kill)break;

      volatile JobPointer retval;
      if(threadID<(currentJobs[warpID]->numThreads)) 
          retval = executeJob(currentJobs[warpID]);
      if(*kill)break;

      if(threadID==0) EnqueueResult(retval, results, kill);
    }
}

__device__ JobPointer executeJob(JobPointer currentJob){

  int JobType = currentJob->JobType;

  // large switch
  switch(JobType){
    case 0:
      addSleep(currentJob->params);
      break;
    case 1:
      break;
    case 2:
      break;
    case 3:
      break;
  }
  return currentJob;
}

