#include "gemtc.cu"
#include <hip/hip_runtime.h>

int main(int argc, char **argv){

  // creates two queues each size of half this param
  setupGemtc(2560);

  // doing work on gpu
  int i;
  for(i=0; i<50; i++){
    int sleepTime = 1000;
    // runs a task on the gpu
    void *ret = run(0, 32, &sleepTime, sizeof(int));
    printf("Finished job with parameter: %d\n", *(int *)ret);
  }

  // stops the superkernel and cleans up some memory
  cleanupGemtc();

  return 0;
}
