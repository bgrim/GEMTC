#include "gemtc.cu"
#include <hip/hip_runtime.h>

int main(int argc, char **argv){

  // creates two queues each size of half this param
  setupGemtc(2560);

  // doing work on gpu
  int i;
  for(i=0; i<10; i++){
    int sleepTime = 1;
    // runs a task on the gpu
    void *ret = run(0, 32, &sleepTime, sizeof(int));
    //printf("%d : Finished job with parameter: %d\n",i, *(int *)ret);
  }

  // stops the superkernel and cleans up some memory
  cleanupGemtc();

  return 0;
}
