#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Kernels/AddSleep.cu"
#include "Kernels/VecAdd.cu"
#include "Kernels/VecDot.cu"
#include "Kernels/MatrixSquare.cu"
#include "Kernels/MatrixMultiply.cu"
#include "Kernels/MatrixTranspose.cu"
#include "Kernels/MatrixVector.cu"
#include "Kernels/MatrixInverse.cu"
#include "Kernels/StencilCopy.cu"
#include "Kernels/StencilUpdate.cu"
#include "Kernels/BlackScholes.cu"


__device__ JobPointer executeJob(volatile JobPointer currentJob);

__global__ void superKernel(volatile Queue incoming, 
                            volatile Queue results, volatile int *kill)
{ 
    // init and result are arrays of integers where result should end up
    // being the result of incrementing all elements of init.
    // They have n elements and are (n+1) long. The should wait for the
    // first element to be set to zero
    int warp_size = 32;

    int threadID = threadIdx.x % warp_size;
    int warpID = threadIdx.x / warp_size;   //added depenency on block

    __shared__ JobPointer currentJobs[32];

    while(!(*kill))
    {
      if(threadID==0)
          FrontAndDequeueJob(incoming, &currentJobs[warpID], kill);
      if(*kill)break;

      volatile JobPointer retval;
      if(threadID<(currentJobs[warpID]->numThreads)) 
          retval = executeJob(currentJobs[warpID]);
      if(*kill)break;

      if(threadID==0) EnqueueResult(retval, results, kill);
    }
}

__device__ JobPointer executeJob(JobPointer currentJob){

  int JobType = currentJob->JobType;

  // large switch
  switch(JobType){
    case 0:
      addSleep(currentJob->params);
      break;
    case 1:
      VecAdd(currentJob->params);
      break;
    case 2:
      MatrixSquare(currentJob->params);
      break;
    case 3:
      VecDot(currentJob->params);
      break;
    case 4:
      MatrixMultiply(currentJob->params);
      break;
    case 5:
      MatrixTranspose(currentJob->params);
      break;
    case 6:
      MatrixVector(currentJob->params);
      break;
    case 7:
      MatrixInverse(currentJob->params);
      break;
    case 8:
      StencilCopy(currentJob->params);
    case 9:
      StencilUpdate(currentJob->params);
      break;
    case 10:
      BlackScholes(currentJob->params);
      break;
  }
  return currentJob;
}

