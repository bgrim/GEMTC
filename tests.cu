#include "hip/hip_runtime.h"
#include "allocations.cu"

void testSleep()
{
    int sleepTime = 5;
    void* ret = run(0, 32, &sleepTime, sizeof(int));
    free(ret);
}
void testAdd()
{
    void* param; void* ret;
    // runs a task on the gpu
    int size = sizeof(float)*(32*3+1);
    param = makeVectorAddArgs(64, size);
    ret = run(1, 32, param, size);
    free(ret);free(param);
}

void testVectorProduct()
{
    int size = sizeof(float)*(32*3+1);
    void* param = makeVectorAddArgsFloat(size);
    void* ret = run(3, 32, param, size);
    free(ret);free(param);
}

void testMatrixSquare()
{
    int ROW = 32;
    int size = 0;
    void* param = makeMatrixTranspose(ROW, size);
    void* ret = run(2, 32, param,size);
    free(ret);free(param);
}
    
void testMatrixMultiply()
{
    int ROW = 32;
    int size = 0;
    void* param = makeMatrixMult(ROW, size);
    void* ret = run(4, 32, param,size);
    free(ret);free(param);
}

void testMatrixTranspose()
{
    int ROW = 32; int COLUMN = 32; int number = 1;
    int size = (number*ROW*COLUMN);
    void* param = makeMatrixTranspose(ROW, size);
    void* ret = run(5, 32, param,size);
    free(ret);free(param);
}

void testMatrixVector()
{
    int ROW = 512; int size = 0;
    void* param = makeMatrixVectorArgs(ROW, size);
    void* ret = run(6, 32, param,size);
    free(ret);free(param);
}

void testMatrixInverse()
{
    int ROW = 3; int size = 0;
    void* param = makeMatrixInverse(ROW, size);
    void* ret = run(7, 32, param,size);
    free(ret);free(param);
}

void testStencil()
{
   int N  =  128;
   float dt = 0.00001f;
   float time = 0.4f;
   int step = ceil(time/dt);
   int size = 0;
   void* param = allocateStencil( N,size);
   step = 2;
   for (int t=0; t<step; t++)
   {    
        void* ret = run(8, 32, param,size);
        free(ret);
        ret = run(9, 32, param,size);
        free(ret);
        printf("Processing time step: %d\n", t);
   }
   free(param);
}
