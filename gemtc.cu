#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

hipStream_t stream_dataIn, stream_dataOut, stream_kernel;

pthread_mutex_t memcpyLock;
pthread_mutex_t enqueueLock;
pthread_mutex_t dequeueLock;

int *d_kill;

#include "DataMovement.cu"
#include "malloc/GemtcMalloc.cu"
#include "Queues/QueueJobs.cu"
#include "SuperKernel.cu"

Queue d_newJobs, d_finishedJobs;
/////////////////////
//Utility Functions//
/////////////////////
void *moveToCuda(void *val, int size){
  void *ret = gemtcMalloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "in moveToCuda of run()");
  return ret;
}
void *moveFromCuda(void *val, int size){
  void *ret = malloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "in moveFromCuda of run()");
  return ret;
}

/////////////////
//API Functions//
/////////////////
void setupGemtc(int QueueSize){
  pthread_mutex_init(&memcpyLock, NULL);
  pthread_mutex_init(&enqueueLock, NULL);
  pthread_mutex_init(&dequeueLock, NULL);

  int warp_size = 32;

  int warps = 16;
  int blocks = 7;

  dim3 threads(warp_size*warps, 1, 1);
  dim3 grid(blocks, 1, 1);

  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);
  
  d_newJobs = CreateQueue(QueueSize);
  d_finishedJobs = CreateQueue(QueueSize);

  int temp = 0;
  d_kill = (int *) moveToCuda((void *)&temp, sizeof(int));

  hipDeviceSynchronize();

//Launch the super kernel
  superKernel<<< grid, threads, 0, stream_kernel>>>
             (d_newJobs, d_finishedJobs, d_kill);
}


int ID =0;
void *run(int Type, int Threads, void *host_params, int size_params){
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = Type;
  h_JobDescription->numThreads = Threads;
  h_JobDescription->params = moveToCuda(host_params, size_params);

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section
  int MyID = ID++;
  h_JobDescription->JobID = MyID;

  EnqueueJob(h_JobDescription, d_newJobs);
  pthread_mutex_unlock(&enqueueLock); //End Critical Section
  printf("Finished enqueue #%d\n", MyID);

  int first = 1;
  while(h_JobDescription->JobID!=MyID || first){
    pthread_yield();
    pthread_mutex_lock(&dequeueLock);
    h_JobDescription = FrontResult(d_finishedJobs);
    if(h_JobDescription->JobID==MyID)DequeueResult(d_finishedJobs);
    pthread_mutex_unlock(&dequeueLock);
    first = 0;
  }
  printf("Recieved result on Job #%d\n", MyID);
  
  return moveFromCuda(h_JobDescription->params, size_params); 
}

void cleanupGemtc(){
  int temp=1;
  cudaSafeMemcpy(&temp, d_kill, sizeof(int), hipMemcpyHostToDevice, 
                 stream_dataIn, "Writing the kill command to SuperKernel");

  DisposeQueue(d_newJobs);

  DisposeQueue(d_finishedJobs);

  hipStreamDestroy(stream_kernel);
  hipStreamDestroy(stream_dataIn);
  hipStreamDestroy(stream_dataOut);

  pthread_mutex_destroy(&memcpyLock);
  pthread_mutex_destroy(&enqueueLock);
  pthread_mutex_destroy(&dequeueLock);
}
