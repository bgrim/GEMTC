#include <stdlib.h>
#include "QueueHelpers.cu"

////////////////////////////////////////////////////////////
// Constructor and Deconsturctor
////////////////////////////////////////////////////////////

Queue CreateQueue(int MaxElements) {
  Queue Q = (Queue) malloc (sizeof(struct QueueRecord));

  Q->Array = (JobPointer *) gemtcMalloc(sizeof(JobPointer)*MaxElements);

  Q->Capacity = MaxElements;
  Q->Front = 1;
  Q->Rear = 0;
  Q->ReadLock = 0;

  Queue d_Q = (Queue) gemtcMalloc(sizeof(struct QueueRecord));
  cudaSafeMemcpy(d_Q, Q, sizeof(struct QueueRecord), 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "Copying initial queue to device");
  free(Q);
  return d_Q;
}

void DisposeQueue(Queue d_Q) {
  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  cudaSafeMemcpy(h_Q, d_Q, sizeof(struct QueueRecord), 
                 hipMemcpyDeviceToHost, stream_dataIn,
                 "DisposeQueue, Copying Queue to get array pointer");
  gemtcFree((void *)h_Q->Array);
  free(h_Q);
  gemtcFree(d_Q);
}

////////////////////////////////////////////////////////////
// Host Functions to Change Queues
////////////////////////////////////////////////////////////

void EnqueueJob(JobPointer h_JobDescription, Queue Q) {
//called by CPU

  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataIn,
                 "EnqueueJob, Getting Queue");

  while(h_IsFull(h_Q)){
    pthread_yield();
    cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataIn,
                    "EnqueueJob, Getting Queue again...");
  }

  // floating point exception from mod capacity if 0 or -n
  h_Q->Rear = (h_Q->Rear+1)%(h_Q->Capacity);

  JobPointer d_JobDescription = 
      (JobPointer) gemtcMalloc(sizeof(struct JobDescription));

  cudaSafeMemcpy( d_JobDescription,
                  h_JobDescription, 
                  sizeof(struct JobDescription),
                  hipMemcpyHostToDevice, 
                  stream_dataIn,
                  "EnqueueJob, Writing JobDescription");

  // set job description
  cudaSafeMemcpy( (void *)&h_Q->Array[h_Q->Rear],
                  &d_JobDescription, 
                  sizeof(JobPointer),
                  hipMemcpyHostToDevice, 
                  stream_dataIn,
                  "EnqueueJob, Writing JobPointer");

  cudaSafeMemcpy(movePointer(Q, 12), movePointer(h_Q, 12), 
		 sizeof(int), hipMemcpyHostToDevice, stream_dataIn,
                 "EnqueueJob, Updating Queue");
  free(h_Q);
}

JobPointer FrontResult(Queue Q) {
//called by CPU
  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));

  cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting Queue");
  while(h_IsEmpty(h_Q)){
    pthread_yield();
    cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                   "FandDJob, Getting Queue again...");
  }
  JobPointer *resultP = (JobPointer *) malloc(sizeof(JobPointer));
  JobPointer result = (JobPointer) malloc(sizeof(struct JobDescription));

  cudaSafeMemcpy(resultP, (void *)&h_Q->Array[h_Q->Front], sizeof(JobPointer), 
                 hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting JobPointer");

  cudaSafeMemcpy(result, (void *)*resultP, sizeof(struct JobDescription), 
                 hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting JobDescription");

  free(h_Q);
  gemtcFree(*resultP);
  free(resultP);

  return result;
}
void DequeueResult(Queue Q) {
//called by CPU
  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));

  cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting Queue");

  while(h_IsEmpty(h_Q)){
    pthread_yield();
    cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                   "FandDJob, Getting Queue again...");
  }

  h_Q->Front = (h_Q->Front+1)%(h_Q->Capacity);

  cudaSafeMemcpy( movePointer(Q, 16), movePointer(h_Q, 16), 
		  sizeof(int), hipMemcpyHostToDevice, stream_dataOut,
                  "FandDJob, Updating Queue");

  free(h_Q);
}


////////////////////////////////////////////////////////////
// Device Functions to Change Queues
////////////////////////////////////////////////////////////
__device__ void FrontAndDequeueJob(volatile Queue Q, volatile JobPointer *pResult, 
                                   volatile int *kill) {
//called by GPU
  getLock(Q);

  int count = 0;
  while(d_IsEmpty(Q)){
    if(*kill)return;
    count++;
  }
  volatile int *front = &Q->Front;

  volatile JobPointer *ppResult = Q->Array + *front;

  *pResult = *ppResult;

  *front = (*front+1)%(Q->Capacity);

  releaseLock(Q);
}

__device__ void EnqueueResult(volatile JobPointer X, volatile Queue Q, volatile int *kill) {
//called by GPU
  getLock(Q);

  int count =0;
  while(d_IsFull(Q)){
    count++;
    if(*kill)return;
  }
  volatile int *rear = &Q->Rear;
  int temp = (*rear + 1)%(Q->Capacity);

  volatile JobPointer *pLoc = Q->Array + temp;
  *pLoc = X;

  *rear = temp;

  releaseLock(Q);
}




