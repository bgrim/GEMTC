
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void addSleep(void *p_us_time)
{ 
    //This method will sleep for clockRate*kernel_time many clock ticks
    // which is equivalent to sleeping for kernel_time milliseconds
    int time = *((int *) p_us_time);

    float AddPerUs = 10.26188; //Ben
    //        float AddPerUs = 9.89759943623274; //Scott
    //    float AddPerUs = 1; // Test

    float adds = time*AddPerUs;

    int temp=0;
    while(temp<adds){
       temp++;
    }
}
