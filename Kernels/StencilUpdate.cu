
#include <hip/hip_runtime.h>
// GPU kernel
__device__ void StencilUpdate(void* param)
{
    float* paramIn = (float*)param;
    int N = (int)paramIn[0];
    float h = paramIn[1];
    float dt = paramIn[2];
    float alpha = paramIn[3];
    float* u = paramIn+5;
    float* u_prev = paramIn+5+N*N;
    // Setting up indices
    int i = threadIdx.x;
    int I = i;
    //if (I>=N*N){return;}    
    while (I < N*N)
    {
    // if not boundary do
    if ( (I>N) && (I< N*N-1-N) && (I%N!=0) && (I%N!=N-1)) 
    {    
        u[I] = u_prev[I] + alpha*dt/(h*h) * (u_prev[I+1] + u_prev[I-1] + u_prev[I+N] + u_prev[I-N] - 4*u_prev[I]);
    }
    I = I + 32;
    }
    // Boundary conditions are automatically imposed
    // as we don't touch boundaries
}

