
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void MatrixTranspose(void *input)
{ 
    float* inputIn = (float*)input; 
    int N = (int)inputIn[0];
    float *matrix = inputIn+1;
    float *matrixT = matrix + N*N;
    int warp_size=32;
    int threadX = threadIdx.x % warp_size;
    for (unsigned int i = threadX; i < N; i=i+32)
    {
       //int i = threadX;
       //if (i < N)
       { 
          for (int idx = 0; idx < N; ++idx)
          {
             int idx_in = i*N+idx;
             int idx_out = idx*N+i;
             //printf("%d,%d\n",idx_in,idx_out);
             matrixT[idx_out] = matrix[idx_in]; 
             //printf("%.2f->%.2f\n", matrix[idx_in] , matrixT[idx_out]); 
          }
       }
    }
#if 0 
    if (threadX < 32 && threadY < 32)
    {
       int idx_in = threadX + 32*threadY;
       int idx_out = threadY + 32*threadX;
       matrix[idx_out] = matrix[idx_in]; 
    }
#endif
}
