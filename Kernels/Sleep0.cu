
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void sleep0(void *p_kernel_time, int clockRate)
{ 
    //This method will sleep for clockRate*kernel_time many clock ticks
    // which is equivalent to sleeping for kernel_time milliseconds
    int kernel_time = *((int *) p_kernel_time);

    int finish_clock;
    int start_time;
    for(int temp=0; temp<kernel_time; temp++){
        start_time = clock();
        finish_clock = start_time + clockRate;
        bool wrapped = finish_clock < start_time;
        while( clock() < finish_clock || wrapped) wrapped = clock()>0 && wrapped;
    }
}
