
#include <hip/hip_runtime.h>
__device__ void VecDot( void* param)
{
    int N = 32;
    float* paramIn = (float*)param;
    float* paramInOrig = (float*)param;
    int size = (int)(*paramIn);
    paramIn = paramIn + 1;
    float* a  = paramIn;
    paramIn = paramIn + size;
    float* b = paramIn;
    paramIn = paramIn + size;
    float* c = paramIn;
    //int tid = threadIdx.x%N + blockIdx.x * blockDim.x;
    int tid = threadIdx.x%N;
    int cacheIndex = threadIdx.x%N;
    float   temp = 0;
    if (tid < N) 
    {
        temp += a[tid] * b[tid];
        //tid += blockDim.x * gridDim.x;
    }
    // set the cache values
    c[cacheIndex] = temp;
    
#if 0     
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    //int i = blockDim.x/2;
    int i = N/2;
    while (i != 0) {
        if (cacheIndex < i)
            c[cacheIndex] += c[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    __syncthreads();
    if (cacheIndex == 0)
    {
       paramInOrig[0] = 44;
       printf("Val: %d\n", paramInOrig[0]);
    }
#endif
     paramInOrig[0] = 44;
}

#if 0
__global__ void dot( float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}
#endif
