
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void VecAdd ( void* param1)
{
   int* mem = (int*)param1;
   int size = mem[0];
   int* A = mem+1;
   int* B = A+size;
   int* C = B+size;
   int warp_size = 32;
   int tid = threadIdx.x%warp_size;
   //C[tid] = A[tid] + B[tid];
#if 1 
   while (tid < size)
   {
      C[tid] = A[tid] + B[tid];
      //printf("tid:%d, C=%d\n", tid, C[tid]);
      tid = tid + warp_size;
   }
#endif

#if 0 
   int* A = (int*)param1;
   int* B = (int*)param1;
   int* C = (int*)param1;

   int tid = threadIdx.x + blockIdx.x*blockDim.x;

   while (tid < 32) {
      C[tid] = A[tid] + B[tid];
      tid += blockDim.x*gridDim.x;
   }
#endif
}
