
#include <hip/hip_runtime.h>
__device__ void MatrixVector(void* param)
{
    int* paramIn = (int*)param;
    int N = paramIn[0];
    int* A = paramIn+1;
    int* B = paramIn+1+N*N;
    int* C = paramIn+1+N*N+N;
 
    int bx = blockIdx.x; 
    // Assume one block
    bx = 0;
    //int by = blockIdx.y;
    int tx = threadIdx.x%32; 
    // Calculate the row index of the Pd element and M

    int Row = bx * 32 + tx;
  
    for (unsigned int i = Row; i < N; i=i+32)
    {
       //if(i < N)         
       {
          int Pvalue = 0;
          for (unsigned int k = 0; k < N; k++) 
          {
              Pvalue += A[i*N+k] * B[k];
          }
          C[i] = Pvalue;
          //printf("%d=%d\n",i,Pvalue);
       }
    }
}
