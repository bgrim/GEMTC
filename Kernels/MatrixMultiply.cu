
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void MatrixMultiply(void *input)
{ 
    float* inputIn = (float*)input;
    int matrixWidth = inputIn[0];
    float *matrixA = inputIn+1;
    float *matrixB = matrixA + matrixWidth*matrixWidth;
    float *matrixOut = matrixA + 2*matrixWidth*matrixWidth;
    int warp_size=32;
    int thread = threadIdx.x % warp_size;
        
    for (unsigned int i = thread; i < matrixWidth; i=i+32)
    {
      for (unsigned int j = 0; j < matrixWidth; j++) {
         float sum = 0;
         for (unsigned int k = 0; k < matrixWidth; k++) {
           float a = matrixA[i * matrixWidth + k];
           float b = matrixB[k * matrixWidth + j];
           sum += a * b;
         }
         matrixOut[i * matrixWidth + j ] = sum;
      }
   }
}
