
#include <hip/hip_runtime.h>
__device__ void MatrixInverse(  void* param)
{
    float* paramIn = (float*)param;
    int N = (int)paramIn[0];
    paramIn = paramIn+1;
    float* A = paramIn;
    float* B = paramIn+N*N;
    int x = threadIdx.x;
    if (x < N)
    {
   for (int y = 0; y < N; ++y)
   { 
       float pivot = 0;
       for (int i = 0; i < N; ++i)
       {
           pivot = A[N*y+i]/A[N*i+i];
           if (y != i )
           {
               A[N*y+x] = A[N*y+x]-(pivot*A[N*i+x]);
               B[N*y+x] = B[N*y+x]-(pivot*B[N*i+x]);
           }
       }
   }
   for (int y = 0; y < N; ++y)
   {
      for (int i = 0; i < N; ++i)
      {
          if (y == i)
          {
              B[N*y+x] /= A[N*y+y];
              float div = A[N*y+y];
              A[N*y+y] /= div; 
          }
       }
   }
   }
}
