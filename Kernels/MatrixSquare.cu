
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void matrixSquare(void *input)
{ 
    float *matrix = (float *) input;
    int warp_size=32;
    int thread = threadIdx.x % warp_size;
        
    int matrixWidth = 32;
    for (unsigned int i = thread; i < matrixWidth; i=i+32)
    {
      for (unsigned int j = 0; j < matrixWidth; j++) {
         float sum = 0;
         for (unsigned int k = 0; k < matrixWidth; k++) {
           float a = matrix[i * matrixWidth + k];
           float b = matrix[k * matrixWidth + j];
           sum += a * b;
         }
         matrix[i * matrixWidth + j + (matrixWidth * matrixWidth)] = sum;
      }
   }
}
