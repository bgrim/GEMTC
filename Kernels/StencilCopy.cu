
#include <hip/hip_runtime.h>
__device__ void StencilCopy(void* param)
{
    float* paramIn = (float*)param;
    int N = (int)paramIn[0];
    float* u = paramIn+5;
    float* u_prev = paramIn+5+N*N;
    int i = threadIdx.x;
    int I = i;
    while (I < N*N)
    {
        //if (I>=N*N){return;}    
        u_prev[I] = u[I];
        I = I + 32;
    }
}

