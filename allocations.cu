
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
int test(int a, int b)
{
   printf(">>%d, %d\n", a, b);
   return 0;
}


void* makeVectorAddArgsFloat(int size)
{
   float* mem = (float*)malloc(size);
   float* a1 = mem+1;
   float* b1 = a1+32;
   float* c1 = b1+32;
   for (int idx = 0; idx < 32; ++idx)
   {
      a1[idx] = idx;
      b1[idx] = idx;
      c1[idx] = 0;
   }
   mem[0] = 32;
   return (void*)mem;
}

void* makeVectorAddArgs(int N, int & size)
{
   size = (3*N+1)*sizeof(int);
   int* mem = (int*)malloc(size);
   int* a1 = mem+1;
   int* b1 = a1+N;
   int* c1 = b1+N;
   for (int idx = 0; idx < N; ++idx)
   {
      a1[idx] = idx;
      b1[idx] = idx;
      c1[idx] = 0;
   }
   mem[0] = N;
   return (void*)mem;
}


float *makeMatrixTranspose(int ROW, int& size)
{
  int COLUMN = ROW;

  int a=0, b=0;
  size = (2*ROW*ROW+1)*sizeof(float);
  float *stuff = (float *) malloc(size);
  stuff[0] = ROW;
  float* matrixIn = stuff+1;
  float* matrixOut = matrixIn + ROW*ROW;
  for(a=0; a<ROW;a++)
  {
      for(b=0; b<COLUMN;b++)
      {
         //matrix[b + a * ROW]=((float)rand())/((float) RAND_MAX);
         matrixIn[b + a * ROW]=b;
         matrixOut[b + a * ROW]=0;
      }
  }
  return stuff;
}

float *makeMatrixInverse(int ROW, int& size)
{
    float* stuff = makeMatrixTranspose(ROW, size);
    float* matrixIdent = stuff + 1 + ROW*ROW;
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           if (idx == jdx)
              matrixIdent[idx*ROW+jdx] = 1;
           else
              matrixIdent[idx*ROW+jdx] = 0;
        }
    }
    return stuff;
}


void *makeMatrix(int ROW, int& size)
{
  int COLUMN = ROW;

  int a=0, b=0;
  size = (1+2*ROW*COLUMN)*sizeof(float);
  float *stuff = (float *) malloc(size);
  stuff[0] = ROW;
  for(a=0; a<ROW;a++)
  {
    for(b=0; b<COLUMN;b++)
    {
      stuff[a + b * ROW]=((float)rand())/((float) RAND_MAX);
      stuff[a + b * ROW + ROW * COLUMN] = 0.0;
    }
  }
  return stuff;
}

void* makeMatrixMult(int ROW, int& size)
{
  int COLUMN = ROW;
  int a=0, b=0;
  size = (3*ROW*ROW+1)*sizeof(float);
  float *stuff = (float *) malloc(size);
  float* orig = stuff;
  // first parameter is the matrix size
  *stuff = ROW;
  // increment the pointer by one
   stuff = stuff+1;
  for(a=0; a<ROW;a++)
    {
      for(b=0; b<COLUMN;b++)
      {
         stuff[a + b * ROW]= ((float)rand())/((float) RAND_MAX);
         stuff[a + b * ROW + ROW * COLUMN] = 
                     ((float)rand())/((float) RAND_MAX);
         stuff[a + b * ROW + 2*ROW * COLUMN] = 0.0;
      }
    }
  return orig;
}

void* makeMatrixVectorArgs(int ROWS, int& size)
{
    size = (ROWS*ROWS+2*ROWS+1)*sizeof(int);
    int* param = (int*)malloc(size);
    param[0] = ROWS;
    int* matrix = param+1;
    int* vecA = matrix+ROWS*ROWS;
    int* vecB = vecA+ROWS;
    // idx = row
    for (int idx=0;idx<ROWS;++idx)
    {
        // for each column value, jdx = column
        for (int jdx=0;jdx<ROWS;++jdx)
            matrix[jdx+idx*ROWS]=idx;
        vecA[idx]=idx;
        vecB[idx]=idx;
    }
    return (void*)param;
}

void* allocateStencil(int N, int& size)
{
    float xmin     = 0.0f;
    float xmax     = 3.5f;
    float ymin     = 0.0f;
    //float ymax     = 2.0f;
    float h       = (xmax-xmin)/(N-1);
    float dt    = 0.00001f;    
    float alpha    = 0.645f;
    float time     = 0.4f;

    int steps = ceil(time/dt);
    int I;

    //float *u      = new float[N*N];
    //float *u_host = new float[N*N];

    size = sizeof(float)*(5+2*N*N);
    float* param = (float*)malloc(sizeof(float)*size);
     
    param[0] = N;
    param[1] = h;
    param[2] = dt;
    param[3] = alpha;
    param[4] = N;
    float* u = param+5;
    float* u_host = u + N*N;
    // Generate mesh and intial condition
    for (int j=0; j<N; j++)
    {    for (int i=0; i<N; i++)
        {    I = N*j + i;
            u[I] = 0.0f;
            u_host[I] = 0.0f;
            if ( (i==0) || (j==0)) 
                {u[I] = 200.0f;}
        }
    }

    return (void*)param;
}
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}
void* allocateBlackScholes(int N, int& size)
{
    const int   OPT_N = N;//4000000;
    const int   OPT_SZ = OPT_N * sizeof(float);
    const float RISKFREE = 0.02f;
    const float VOLATILITY = 0.30f;
    float *h_CallResultCPU, *h_PutResultCPU;
    float *h_StockPrice, *h_OptionStrike, *h_OptionYears;


    size = OPT_SZ*5+3*sizeof(float);
    
    float* param = (float*)malloc(size);
    param[0] = RISKFREE;
    param[1] = VOLATILITY;
    param[2] = OPT_N;
    h_CallResultCPU = param+3;
    h_PutResultCPU = h_CallResultCPU + OPT_N;
    h_StockPrice = h_PutResultCPU + OPT_N;
    h_OptionStrike = h_StockPrice + OPT_N;
    h_OptionYears = h_OptionStrike + OPT_N;
    srand(5347);
    //Generate options set
    for(int i = 0; i < OPT_N; i++)
    {
        h_CallResultCPU[i] = 0.0f;
        h_PutResultCPU[i]  = -1.0f;
        h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
        h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
        h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
   }
   return (void*)param;  
}
