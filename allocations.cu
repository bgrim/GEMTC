
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
int test(int a, int b)
{
   printf(">>%d, %d\n", a, b);
   return 0;
}


void* makeVectorAddArgsFloat(int size)
{
   float* mem = (float*)malloc(size);
   float* a1 = mem+1;
   float* b1 = a1+32;
   float* c1 = b1+32;
   for (int idx = 0; idx < 32; ++idx)
   {
      a1[idx] = idx;
      b1[idx] = idx;
      c1[idx] = 0;
   }
   mem[0] = 32;
   return (void*)mem;
}

void* makeVectorAddArgs(int N, int & size)
{
   size = (3*N+1)*sizeof(int);
   int* mem = (int*)malloc(size);
   int* a1 = mem+1;
   int* b1 = a1+N;
   int* c1 = b1+N;
   for (int idx = 0; idx < N; ++idx)
   {
      a1[idx] = idx;
      b1[idx] = idx;
      c1[idx] = 0;
   }
   mem[0] = N;
   return (void*)mem;
}


float *makeMatrixTranspose(int ROW, int& size)
{
  int COLUMN = ROW;

  int a=0, b=0;
  size = (2*ROW*ROW+1)*sizeof(float);
  float *stuff = (float *) malloc(size);
  stuff[0] = ROW;
  float* matrixIn = stuff+1;
  float* matrixOut = matrixIn + ROW*ROW;
  for(a=0; a<ROW;a++)
  {
      for(b=0; b<COLUMN;b++)
      {
         //matrix[b + a * ROW]=((float)rand())/((float) RAND_MAX);
         matrixIn[b + a * ROW]=b;
         matrixOut[b + a * ROW]=0;
      }
  }
  return stuff;
}

float *makeMatrixInverse(int ROW, int& size)
{
    float* stuff = makeMatrixTranspose(ROW, size);
    float* matrixIdent = stuff + 1 + ROW*ROW;
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           if (idx == jdx)
              matrixIdent[idx*ROW+jdx] = 1;
           else
              matrixIdent[idx*ROW+jdx] = 0;
        }
    }
    return stuff;
}


void *makeMatrix(int ROW, int& size)
{
  int COLUMN = ROW;

  int a=0, b=0;
  size = (1+2*ROW*COLUMN)*sizeof(float);
  float *stuff = (float *) malloc(size);
  stuff[0] = ROW;
  for(a=0; a<ROW;a++)
  {
    for(b=0; b<COLUMN;b++)
    {
      stuff[a + b * ROW]=((float)rand())/((float) RAND_MAX);
      stuff[a + b * ROW + ROW * COLUMN] = 0.0;
    }
  }
  return stuff;
}

void* makeMatrixMult(int ROW, int& size)
{
  int COLUMN = ROW;
  int a=0, b=0;
  size = (3*ROW*ROW+1)*sizeof(float);
  float *stuff = (float *) malloc(size);
  float* orig = stuff;
  // first parameter is the matrix size
  *stuff = ROW;
  // increment the pointer by one
   stuff = stuff+1;
  for(a=0; a<ROW;a++)
    {
      for(b=0; b<COLUMN;b++)
      {
         stuff[a + b * ROW]= ((float)rand())/((float) RAND_MAX);
         stuff[a + b * ROW + ROW * COLUMN] = 
                     ((float)rand())/((float) RAND_MAX);
         stuff[a + b * ROW + 2*ROW * COLUMN] = 0.0;
      }
    }
  return orig;
}

void* makeMatrixVectorArgs(int ROWS, int& size)
{
    size = (ROWS*ROWS+2*ROWS+1)*sizeof(int);
    int* param = (int*)malloc(size);
    param[0] = ROWS;
    int* matrix = param+1;
    int* vecA = matrix+ROWS*ROWS;
    int* vecB = vecA+ROWS;
    // idx = row
    for (int idx=0;idx<ROWS;++idx)
    {
        // for each column value, jdx = column
        for (int jdx=0;jdx<ROWS;++jdx)
            matrix[jdx+idx*ROWS]=idx;
        vecA[idx]=idx;
        vecB[idx]=idx;
    }
    return (void*)param;
}
