
#include <hip/hip_runtime.h>
#include <stdio.h>

//__device__ int waitForValue(int *flag);

__device__ void clock_block(int kernel_time, int clockRate)
{ 
    int finish_clock;
    int start_time;
    for(int temp=0; temp<kernel_time; temp++){
        start_time = clock();
        finish_clock = start_time + clockRate;
        bool wrapped = finish_clock < start_time;
        while( clock() < finish_clock || wrapped) wrapped = clock()>0 && wrapped;
    }
}

__device__ int waiting(volatile int *temp){
  return *temp==0;
}

__global__ void superKernel(int *d_flagIn, int *d_flagOut)
{ 
    // init and result are arrays of integers where result should end up
    // being the result of incrementing all elements of init.
    // They have n elements and are (n+1) long. The should wait for the
    // first element to be set to zero
    int threadID = (threadIdx.x + threadIdx.y * blockDim.x);
    
    //clock_block(10,1000000);

    //int count = waitForValue(d_flagIn);

    volatile int *temp = (volatile int *)d_flagIn;
    int count=0;
    while(waiting(temp)){ 
      count++; 
    }

    if(threadID==0) *d_flagOut = count;
}

/*
__device__ int waitForValue(int *flag){
   int count = 0;
   while(true){ 
      count++;
      int *temp = (int *) malloc(sizeof(int));
      *temp = *flag;
      if (*temp!=0) {free(temp); return count;}
      free(temp);
   }
}
*/