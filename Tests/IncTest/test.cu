#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include "Kernels/incSuperKernel.cu"

#include <pthread.h>

/////////////////////////////////////////////////////////////////
// Global Variables
/////////////////////////////////////////////////////////////////

void printAnyErrors()
{
  hipError_t e = hipGetLastError();
  printf("CUDA error:  %s \n", hipGetErrorString(e));
  
}

////////////////////////////////////////////////////////////////////
// The Main
////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
  hipStream_t stream_kernel, stream_dataIn, stream_dataOut;
  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);  //currently these arent used



  int size = 5;

  int* h_init = (int*)malloc((size+1)*sizeof(int));
  int* h_result = (int*)malloc((size+1)*sizeof(int));

  int* d_init;
  hipMalloc(&d_init, (size+1)*sizeof(int));
  int* d_result;
  hipMalloc(&d_result, (size+1)*sizeof(int));

  h_init[0]=0;  //set the data ready flag to false
  hipMemcpyAsync(d_init, h_init, sizeof(int), hipMemcpyHostToDevice,stream_dataIn);
  hipStreamSynchronize(stream_dataIn);

  h_result[0]=0;  //set the data ready flag to false
  hipMemcpyAsync(d_result, h_result, sizeof(int), hipMemcpyHostToDevice,stream_dataOut);
  hipStreamSynchronize(stream_dataOut);

  dim3 threads(32, 1);
  dim3 grid(1, 1);

  printf("launching SuperKernel\n");

// call the cudaMatrixMul cuda function
  superKernel<<< grid, threads, 0, stream_kernel>>>(d_init, size, d_result);

//PRINT HERE
  printAnyErrors();

//Make inputs and transfer them
  int j;
  for(j=1;j<size+1;j++)h_init[j] = j;

  printf("launching hipMemcpy Data\n");

  hipMemcpyAsync(&d_init[1], &h_init[1], size*sizeof(int), hipMemcpyHostToDevice, stream_dataIn);
  hipStreamSynchronize(stream_dataIn); 

//PRINT HERE
  printAnyErrors();

//Mark flag as ready
  printf("launching hipMemcpy Flag\n");

  h_init[0]=7;
  hipMemcpyAsync(d_init, h_init, sizeof(int), hipMemcpyHostToDevice,stream_dataIn);
  hipStreamSynchronize(stream_dataIn);
 

//wait for result flag to be on
  while(h_result[0]==0) { hipMemcpyAsync(h_result, d_result, sizeof(int), hipMemcpyDeviceToHost, stream_dataOut); 
                          hipStreamSynchronize(stream_dataOut); 
                          printf("got value h_result[0]:  %d\n", h_result[0]); }
//PRINT HERE
  printAnyErrors();

//Get and print results
  hipMemcpyAsync(&h_result[1], &d_result[1], size*sizeof(int), hipMemcpyDeviceToHost, stream_dataOut);
  hipStreamSynchronize(stream_dataOut); 
  int i;
  for(i=0; i<size; i++) printf("intial value: %d\t final value: %d\n", h_init[i+1], h_result[i+1]);

//PRINT HERE
  printAnyErrors();

  return 0;    
}







