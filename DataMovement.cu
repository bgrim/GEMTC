

int first = 1;
void synchronizeAndPrint(hipStream_t stream, char *s){
  hipError_t e = hipStreamSynchronize(stream);
  if(e!=hipSuccess){
    //if(first){printf("CUDA Error:   %s   at %s\n", hipGetErrorString( e ), s);first=0;}
    first=0;
    printf("CUDA Error:   %s   at %s\n", hipGetErrorString( e ), s);
  }
}


void cudaSafeMemcpy(void *destination, void *source, int size, enum hipMemcpyKind direction, hipStream_t stream, char *errorStatement)
{
  //Get Lock
  pthread_mutex_lock(&memcpyLock);

  //Memcpy
  hipMemcpyAsync(destination, source, size, direction, stream);

  //Synchronize and Print Errors
  synchronizeAndPrint(stream, errorStatement);

  //Release Lock
  pthread_mutex_unlock(&memcpyLock);
}



