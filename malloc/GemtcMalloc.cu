#include<stdlib.h>
#include<hip/hip_runtime.h>

typedef struct memoryPointer MemoryPointer;

struct memoryPointer{
  MemoryPointer *ptr;
  unsigned size;
  unsigned *data;
};

static MemoryPointer base;
static MemoryPointer *freep = NULL;

int CHUNK_SIZE=256;
int headerSize=16;
int MIN_BULK_AMOUNT = 1000; //2^20 ~1million

void gemtcAddList(MemoryPointer *bp){
  MemoryPointer *p;

  for(p = freep; !(bp->data > p->data && bp->data < (p->ptr)->data); p = p->ptr)
    if(p->data >= (p->ptr)->data && (bp->data > p->data || bp->data < (p->ptr)->data)) 
      break;

  if( (((char *)bp->data) + bp->size) == (char *)p->ptr->data){
    bp->size += (p->ptr)->size;
    bp->ptr = (p->ptr)->ptr;
    cudaSafeMemcpy(bp->data, &bp->size, sizeof(unsigned), hipMemcpyHostToDevice,
                   stream_dataIn, "Merging freed memory in old block");
    if(p->ptr != &base)free(p->ptr);
  }else
    bp->ptr = p->ptr;

  if( (((char *)p->data) + p->size) == (char *)bp->data){
    p->size += bp->size;
    p->ptr = bp->ptr;
    cudaSafeMemcpy(p->data, &p->size, sizeof(unsigned), hipMemcpyHostToDevice,
	       stream_dataIn, "Merging old memory into new block");
    free(bp);
  }else
    p->ptr = bp;
  
  freep = p;
}

void gemtcFree(void *loc){
  loc = ((void *)(((char *)loc)-headerSize));
  MemoryPointer *v = (MemoryPointer *) malloc(sizeof(MemoryPointer));
  cudaSafeMemcpy(&v->size, loc, sizeof(unsigned), hipMemcpyDeviceToHost,
                 stream_dataOut, "Reading size of freed memory");
  v->data = (unsigned *) loc;
  gemtcAddList(v);
}


static MemoryPointer *morecore(unsigned nu){
  void *cp;
  MemoryPointer *up = (MemoryPointer *)malloc(sizeof(MemoryPointer));
  if (nu < MIN_BULK_AMOUNT) nu = MIN_BULK_AMOUNT;
  hipMalloc(&cp, nu);

  up->data = (unsigned *)cp;

  up->size = nu;
  cudaSafeMemcpy(cp,&nu,sizeof(unsigned),hipMemcpyHostToDevice,
		 stream_dataIn, "Writing size of new block from hipMalloc");

  gemtcAddList(up);
  return freep;
}


void *gemtcMalloc(unsigned nbytes){
  MemoryPointer *p, *prevp;
  if ((prevp = freep)==NULL){
    base.ptr = freep = prevp = &base;
    base.size = 0;
  }
  nbytes+=headerSize;
  if(nbytes%CHUNK_SIZE!=0)nbytes+=(CHUNK_SIZE-nbytes%CHUNK_SIZE);
  char *loc;
  for(p = prevp->ptr; ;prevp = p, p = p->ptr){
    if(p->size >= nbytes){
      if(p->size == nbytes){
        prevp->ptr = p->ptr;
        loc = (char *) p->data;
        free(p);
      }else{
        p->size -= nbytes;
        loc =((char *) p->data)+p->size;
      }
      freep = prevp;
      cudaSafeMemcpy(loc,&nbytes,sizeof(unsigned),hipMemcpyHostToDevice,
                     stream_dataIn, "Writing size on newly allocated memory");
      return (void *)(loc+headerSize);
    }
    if (p == freep)
      if((p = morecore(nbytes))==NULL)
        return NULL;
  }
}

